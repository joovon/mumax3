
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

// Calculates the summand F(-k) · [k × F(k)] / k^2
extern "C" __global__ void
solidanglefouriersummand(float* __restrict__  summand_array,
                    float* __restrict__ FkX_array, float* __restrict__ FkY_array, float* __restrict__ FkZ_array,
                    int Nx, int Ny, int Nz) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if(ix>= Nx || iy>= Ny || iz>=Nz) {
        return;
    }

    float kx = static_cast<float>(ix) / Nx;
    float ky = static_cast<float>(iy) / Ny;
    float kz = static_cast<float>(iz) / Nz;

    // Account for positive and negative frequencies (k-space values are in the range [-1/2, 1/2])
    if (ix >= Nx/2)
        kx -= 1.0f;
    if (iy >= Ny/2)
        ky -= 1.0f;
    if (iz >= Nz/2)
        kz -= 1.0f;

    float k2 = kx*kx + ky*ky + kz*kz;

    int I = (iz*Ny + iy)*Nx + ix;
    int e = 2 * I;

    // Avoid division by zero at kx = ky = kz = 0
    if (k2 == 0.0f) {
        summand_array[I] = 0.0f;

    } else {

        float reFkX  =  FkX_array[e  ];
        float reFkY  =  FkY_array[e  ];
        float reFkZ  =  FkZ_array[e  ];
        float imFkX  =  FkX_array[e+1];
        float imFkY  =  FkY_array[e+1];
        float imFkZ  =  FkZ_array[e+1];
        float imFmkX = -FkX_array[e+1];
        float imFmkY = -FkY_array[e+1];
        float imFmkZ = -FkZ_array[e+1];

        hipDoubleComplex FkX  = make_hipDoubleComplex(reFkX, imFkX);
        hipDoubleComplex FkY  = make_hipDoubleComplex(reFkY, imFkY);
        hipDoubleComplex FkZ  = make_hipDoubleComplex(reFkZ, imFkZ);
        hipDoubleComplex FmkX = make_hipDoubleComplex(reFkX, imFmkX);
        hipDoubleComplex FmkY = make_hipDoubleComplex(reFkY, imFmkY);
        hipDoubleComplex FmkZ = make_hipDoubleComplex(reFkZ, imFmkZ);

        hipDoubleComplex kx_comp = make_hipDoubleComplex(kx, 0.0f);
        hipDoubleComplex ky_comp = make_hipDoubleComplex(ky, 0.0f);
        hipDoubleComplex kz_comp = make_hipDoubleComplex(kz, 0.0f);

        // Calculate F(-k) x (k · F(k)) / k^2
        float summand = hipCimag(
                            hipCadd(
                                hipCadd(
                                    hipCmul(FmkX, hipCsub(hipCmul(ky_comp, FkZ), hipCmul(kz_comp, FkY))),
                                    hipCmul(FmkY, hipCsub(hipCmul(kz_comp, FkX), hipCmul(kx_comp, FkZ)))
                                ),
                                hipCmul(FmkZ, hipCsub(hipCmul(kx_comp, FkY), hipCmul(ky_comp, FkX)))
                            )
                        );
        
        summand /= k2;
        summand_array[I] = summand;
    }
}
