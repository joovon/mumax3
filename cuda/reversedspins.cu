#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"

// Simple routine to count the number of spins with mz <= 0
// Used to obtain, e.g., a rough estimate of a skyrmion core size under
// finite temperatures
// See reversedspins.go.
extern "C" __global__ void
setreversedspins(float* __restrict__ s,
                     float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
                     int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz)
    {
        return;
    }

    int I = idx(ix, iy, iz);                      // central cell index

    float3 m0 = make_float3(mx[I], my[I], mz[I]); // +0
    float mz0 = mz[I];
//  int i_;                                       // neighbor index

    if(is0(m0))
    {
        s[I] = 0.0f;
        return;
    }

    if(mz0>0)
    {
        s[I] = 0.0f;
    }
    else
    {
        s[I] = -1.0f;
    }

}
